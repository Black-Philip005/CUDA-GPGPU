#include "hip/hip_runtime.h"
/*
* TP 1 - Premiers pas en CUDA
* --------------------------
* Ex 2: Addition de vecteurs
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{
	__global__ void sumArraysCUDA(const int n, const int *const dev_a, const int *const dev_b, int *const dev_res)
	{
		auto division = n/blockDim.x+1;
		for(int i = 0; i < division; i++)
		{
			dev_res[i+threadIdx.x*division] = dev_a[i+threadIdx.x*division]+dev_b[i+threadIdx.x*division];
		}

	}

    void studentJob(const int size, const int *const a, const int *const b, int *const res)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		int *dev_a = NULL;
		int *dev_b = NULL;
		int *dev_res = NULL;
		
		// Allocate arrays on device (input and ouput)
		const size_t bytes = size * sizeof(int);

		std::cout 	<< "Allocating input (3 arrays): " 
					<< ( ( 3 * bytes ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();
		
		hipMalloc(&dev_a, bytes);
		hipMalloc(&dev_b, bytes);
		hipMalloc(&dev_res, bytes);

		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input arrays) 
    	hipMemcpy(dev_a,a,bytes,hipMemcpyHostToDevice);
    	hipMemcpy(dev_b,b,bytes,hipMemcpyHostToDevice);
		// Launch kernel
		sumArraysCUDA<<<1, 1024>>>(size, dev_a, dev_b, dev_res);

		// Copy data from device to host (output array)  
		hipMemcpy(res,dev_res,bytes,hipMemcpyDeviceToHost);

		// Free arrays on device
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_res);
	}
}

